#include "hip/hip_runtime.h"
#ifndef CUDA_GENERAL_COAL_CU
#define CUDA_GENERAL_COAL_CU

#include <hiprand.h>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>

#include "n_body_sim_cuda.cuh"

// macro for error-handling
#define gpuErrChk(ans) { gpuAssert((ans), (char*)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// Flag for pingpong;
int pingpong = 0;

// Number particles; determined at runtime.
int num_particles;    
 
// Parameters for the kernel.
int num_blocks;
int num_threads_per_block;

// Device buffer variables.
float* particle_vels[2]; 
float* particle_data[3]; 

// Used for PXPOPT algorithm.
#ifdef USE_ACCEL_ARRAY
  float *accel;
#endif

__global__
void cudaInitKernel(float * vels_buffer, float * data_buffer1, float * data_buffer2, float * random, float box_width, 
                    float box_height, float min_vel, float max_vel, int num_particles)
{
  float *vels_x, *vels_y;
  float *pos_x, *pos_y;
  float *mass1, *mass2;

  vels_x = vels_buffer;
  vels_y = vels_buffer + num_particles;
  pos_x = data_buffer1;
  pos_y = data_buffer1 + num_particles;

  mass1 = data_buffer1 + 2 * num_particles;
  mass2 = data_buffer2 + 2 * num_particles;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < num_particles)
  {
    int idx = 4 * i;
    // Randomly initialize velocities and positions. 
    vels_x[i] = min_vel + random[idx] * (max_vel - min_vel);
    vels_y[i] = min_vel + random[idx + 1] * (max_vel - min_vel);
    pos_x[i] = random[idx + 2] * box_width;
    pos_y[i] = random[idx + 3] * box_height;

    // Set mass to 1. 
    mass1[i] = 1;
    mass2[i] = 1;    

    i += blockDim.x * gridDim.x;
  }
}

void alloc_data() {
  // instantiate particle_vels, particle_data on GPU
  gpuErrChk(hipMalloc((void **) &particle_vels[0], sizeof(float) * 2 * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_vels[1], sizeof(float) * 2 * num_particles));
  
  gpuErrChk(hipMalloc((void **) &particle_data[0], sizeof(float) * 3 * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_data[1], sizeof(float) * 3 * num_particles));

  #ifdef USE_ACCEL_ARRAY
    gpuErrChk(hipMalloc((void **) &accel, 2 * sizeof(float) * num_particles));
  #endif
}

void init_data(int h_num_particles, float box_width, float box_height, float min_vel, 
               float max_vel, int h_num_blocks, int h_num_threads_per_block) 
{
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  // instantiate particle_vels, particle_data on GPU
  alloc_data();
   
  // set initial values for particle_vels, particle_data on GPU
  float * random;
  gpuErrChk(hipMalloc((void **) &random, sizeof(float) * num_particles * 4));   
  
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniform(gen, random, num_particles * 4);

  cudaInitKernel<<<num_blocks, num_threads_per_block>>>(particle_vels[0], particle_data[0], particle_data[1], 
                                                        random, box_width, box_height, min_vel, max_vel, num_particles);

  hiprandDestroyGenerator(gen);
  gpuErrChk(hipFree(random));
}
void init_data(int h_num_particles, float *h_particle_data, float *h_particle_vels, int h_num_blocks, int h_num_threads_per_block) {
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  alloc_data();

  // Rearrange data from the original interleaved format to allow coalesced memory accesses.
  float *temp_particle_data, *temp_particle_vels;
  temp_particle_data = new float[num_particles * 3];
  temp_particle_vels = new float[num_particles * 2];
  for (int i = 0; i < num_particles; i++) {
    temp_particle_data[i] = h_particle_data[3 * i];
    temp_particle_data[i + num_particles] = h_particle_data[3 * i + 1];
    temp_particle_data[i + 2 * num_particles] = h_particle_data[3 * i + 2];
    temp_particle_vels[i] = h_particle_vels[2 * i];
    temp_particle_vels[i + num_particles] = h_particle_vels[2 * i + 1];
  }
  gpuErrChk(hipMemcpy(particle_data[0], temp_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_data[1], temp_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_vels[0], temp_particle_vels, 2 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  delete[] temp_particle_data;
  delete[] temp_particle_vels;
}

void delete_data() {
  // free all memory on GPU
  for (int i = 0; i < 2; i++)
  {
    gpuErrChk(hipFree(particle_vels[i]));
    gpuErrChk(hipFree(particle_data[i]));
  }

  #ifdef USE_ACCEL_ARRAY
    gpuErrChk(hipFree(accel));
  #endif
}

__device__
float2 get_accel(float3 pos_data, float * data_old, int num_particles) {
  // sum acceleration from every other particle based on mass, position of both particles
  float2 accel = {0, 0};

  float3 other_data1, other_data2, other_data3, other_data4;
  float x_dist1, x_dist2, x_dist3, x_dist4;
  float y_dist1, y_dist2, y_dist3, y_dist4;

  float *pos_x = data_old;
  float *pos_y = data_old + num_particles;
  float *mass = data_old + 2 * num_particles;

  float accel_mag1, accel_mag2, accel_mag3, accel_mag4;
  for (int i = 0; i < num_particles; i += 4)
  {
    other_data1.x = pos_x[i];
    other_data1.y = pos_y[i];
    other_data1.z = mass[i];
    other_data2.x = pos_x[i+1];
    other_data2.y = pos_y[i+1];
    other_data2.z = mass[i+1];
    other_data3.x = pos_x[i+2];
    other_data3.y = pos_y[i+2];
    other_data3.z = mass[i+2];
    other_data4.x = pos_x[i+3];
    other_data4.y = pos_y[i+3];
    other_data4.z = mass[i+3];

    x_dist1 = pos_data.x - other_data1.x;
    y_dist1 = pos_data.y - other_data1.y;
    x_dist2 = pos_data.x - other_data2.x;
    y_dist2 = pos_data.y - other_data2.y;
    x_dist3 = pos_data.x - other_data3.x;
    y_dist3 = pos_data.y - other_data3.y;
    x_dist4 = pos_data.x - other_data4.x;
    y_dist4 = pos_data.y - other_data4.y;

    accel_mag1 = other_data1.z * pow(x_dist1 * x_dist1 + y_dist1 * y_dist1 + SOFT_FACTOR, -1.5f);
    accel_mag2 = other_data2.z * pow(x_dist2 * x_dist2 + y_dist2 * y_dist2 + SOFT_FACTOR, -1.5f);
    accel_mag3 = other_data3.z * pow(x_dist3 * x_dist3 + y_dist3 * y_dist3 + SOFT_FACTOR, -1.5f);
    accel_mag4 = other_data4.z * pow(x_dist4 * x_dist4 + y_dist4 * y_dist4 + SOFT_FACTOR, -1.5f);

    accel.x -= fma(x_dist1, accel_mag1, fma(x_dist2, accel_mag2, fma(x_dist3, accel_mag3, x_dist4 * accel_mag4)));
    accel.y -= fma(y_dist1, accel_mag1, fma(y_dist2, accel_mag2, fma(y_dist3, accel_mag3, y_dist4 * accel_mag4)));
  }
  return accel;  
}

void get_particle_data(float * h_particle_data, float * h_particle_vels) {
  // copy GPU data into particle_data, particle_vels array
  float *temp_particle_data, *temp_particle_vels;
  temp_particle_data = new float[num_particles * 3];
  temp_particle_vels = new float[num_particles * 2];
  gpuErrChk(hipMemcpy(temp_particle_data, particle_data[1 - pingpong], sizeof(float) * 3 * num_particles, hipMemcpyDeviceToHost));
  gpuErrChk(hipMemcpy(temp_particle_vels, particle_vels[1 - pingpong], sizeof(float) * 2 * num_particles, hipMemcpyDeviceToHost));

  for (int i = 0; i < num_particles; i++) {
    h_particle_data[3 * i] = temp_particle_data[i];
    h_particle_data[3 * i + 1] = temp_particle_data[i + num_particles];
    h_particle_data[3 * i + 2] = temp_particle_data[i + 2 * num_particles];
    h_particle_vels[2 * i] = temp_particle_vels[i];
    h_particle_vels[2 * i + 1] = temp_particle_vels[i + num_particles];
  }
  delete[] temp_particle_data;
  delete[] temp_particle_vels;
}

#endif // CUDA_GENERAL_COAL_CU
