#include "hip/hip_runtime.h"
#ifndef CUDA_GENERAL_COAL_CU
#define CUDA_GENERAL_COAL_CU

#include <hiprand.h>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>

#include "n_body_sim_cuda.cuh"

// macro for error-handling
#define gpuErrChk(ans) { gpuAssert((ans), (char*)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// Flag for pingpong;
int pingpong = 0;

// Number particles; determined at runtime.
int num_particles;    
 
int num_blocks;
int num_threads_per_block;

// Device buffer variables
float* particle_vels[2]; 
float* particle_data[3]; 

#ifdef USE_FORCES_ARRAY
  float *forces;
#endif

__global__
void cudaInitKernel(float * vels_buffer, float * data_buffer1, float * data_buffer2, float * random, float box_width, 
                    float box_height, float min_vel, float max_vel, int num_particles)
{
  float *vels_x, *vels_y;
  float *pos_x, *pos_y;
  float *mass1, *mass2;

  vels_x = vels_buffer;
  vels_y = vels_buffer + num_particles;
  pos_x = data_buffer1;
  pos_y = data_buffer1 + num_particles;

  mass1 = data_buffer1 + 2 * num_particles;
  mass2 = data_buffer2 + 2 * num_particles;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < num_particles)
  {
    int idx = 4 * i;
    // Randomly initialize velocities and positions. 
    vels_x[i] = min_vel + random[idx] * (max_vel - min_vel);
    vels_y[i] = min_vel + random[idx + 1] * (max_vel - min_vel);
    pos_x[i] = random[idx + 2] * box_width;
    pos_y[i] = random[idx + 3] * box_height;

    // Set mass to 1. 
    mass1[i] = 1;
    mass2[i] = 1;    

    i += blockDim.x * gridDim.x;
  }
}

void alloc_data() {
  // instantiate particle_vels, particle_data on GPU
  gpuErrChk(hipMalloc((void **) &particle_vels[0], sizeof(float) * 2 * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_vels[1], sizeof(float) * 2 * num_particles));
  
  gpuErrChk(hipMalloc((void **) &particle_data[0], sizeof(float) * 3 * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_data[1], sizeof(float) * 3 * num_particles));

  #ifdef USE_FORCES_ARRAY
    gpuErrChk(hipMalloc((void **) &forces, 2 * sizeof(float) * num_particles));
  #endif
}

void init_data(int h_num_particles, float box_width, float box_height, float min_vel, 
               float max_vel, int h_num_blocks, int h_num_threads_per_block) 
{
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  // instantiate particle_vels, particle_data on GPU
  alloc_data();
   
  // set initial values for particle_vels, particle_data on GPU
  float * random;
  gpuErrChk(hipMalloc((void **) &random, sizeof(float) * num_particles * 4));   
  
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniform(gen, random, num_particles * 4);

  cudaInitKernel<<<num_blocks, num_threads_per_block>>>(particle_vels[0], particle_data[0], particle_data[1], 
                                                        random, box_width, box_height, min_vel, max_vel, num_particles);

  hiprandDestroyGenerator(gen);
  gpuErrChk(hipFree(random));
}
void init_data(int h_num_particles, float *h_particle_data, float *h_particle_vels, int h_num_blocks, int h_num_threads_per_block) {
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  alloc_data();

  // Rearrange data from the original interleaved format, to allow coalesced memory accesses
  float *temp_particle_data, *temp_particle_vels;
  temp_particle_data = new float[num_particles * 3];
  temp_particle_vels = new float[num_particles * 2];
  for (int i = 0; i < num_particles; i++) {
    temp_particle_data[i] = h_particle_data[3 * i];
    temp_particle_data[i + num_particles] = h_particle_data[3 * i + 1];
    temp_particle_data[i + 2 * num_particles] = h_particle_data[3 * i + 2];
    temp_particle_vels[i] = h_particle_vels[2 * i];
    temp_particle_vels[i + num_particles] = h_particle_vels[2 * i + 1];
  }
  gpuErrChk(hipMemcpy(particle_data[0], temp_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_data[1], temp_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_vels[0], temp_particle_vels, 2 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  delete[] temp_particle_data;
  delete[] temp_particle_vels;
}

void delete_data() {
  // free all memory on GPU
  for (int i = 0; i < 2; i++)
  {
    gpuErrChk(hipFree(particle_vels[i]));
    gpuErrChk(hipFree(particle_data[i]));
  }

  #ifdef USE_FORCES_ARRAY
    gpuErrChk(hipFree(forces));
  #endif
}

__device__
float2 get_force(float3 pos_data, float * data_old, int num_particles) {
  // sum force from every other particle based on mass, position of both particles
  float2 force = {0, 0};

  float3 other_data; // saves about 3s @ 128 threads/block and 1024 particles to store data_old[i], x_dist, and y_dist locally
  float x_dist, y_dist;

  float force_magnitude;
  for (int i = 0; i < num_particles; i++)
  {
    other_data.x = data_old[i];
    other_data.y = data_old[i + num_particles];
    other_data.z = data_old[i + 2 * num_particles];

    x_dist = pos_data.x - other_data.x;
    y_dist = pos_data.y - other_data.y;

    force_magnitude = pos_data.z * other_data.z * pow(x_dist * x_dist + y_dist * y_dist + SOFT_FACTOR, -1.5f);

    force.x -= x_dist * force_magnitude;
    force.y -= y_dist * force_magnitude;
  }
  return force;  
}

void get_particle_data(float * h_particle_data, float * h_particle_vels) {
  // copy GPU data into particle_data, particle_vels array
  float *temp_particle_data, *temp_particle_vels;
  temp_particle_data = new float[num_particles * 3];
  temp_particle_vels = new float[num_particles * 2];
  gpuErrChk(hipMemcpy(temp_particle_data, particle_data[1 - pingpong], sizeof(float) * 3 * num_particles, hipMemcpyDeviceToHost));
  gpuErrChk(hipMemcpy(temp_particle_vels, particle_vels[1 - pingpong], sizeof(float) * 2 * num_particles, hipMemcpyDeviceToHost));

  for (int i = 0; i < num_particles; i++) {
    h_particle_data[3 * i] = temp_particle_data[i];
    h_particle_data[3 * i + 1] = temp_particle_data[i + num_particles];
    h_particle_data[3 * i + 2] = temp_particle_data[i + 2 * num_particles];
    h_particle_vels[2 * i] = temp_particle_vels[i];
    h_particle_vels[2 * i + 1] = temp_particle_vels[i + num_particles];
  }
  delete[] temp_particle_data;
  delete[] temp_particle_vels;
}

#endif // CUDA_GENERAL_COAL_CU
