#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>

#include "n_body_sim_cuda.cuh"

// macro for error-handling
#define gpuErrChk(ans) { gpuAssert((ans), (char*)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// Flag for pingpong;
int pingpong = 0;

// Number particles; determined at runtime.
int num_particles;    
 
int num_blocks;
int num_threads_per_block;

// Algorithm to use.
int algorithm;

// Device buffer variables
float *particle_vels[2]; 
float *particle_data[2];
float *forces;

__global__
void cudaInitKernel(float * vels_buffer, float * data_buffer1, float * data_buffer2, float * random, float box_width, 
                    float box_height, float min_vel, float max_vel, int num_particles)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < num_particles)
  {
    vels_buffer[i] = min_vel + random[4 * i] * (max_vel - min_vel);
    vels_buffer[i + num_particles] = min_vel + random[4 * i + 1] * (max_vel - min_vel);
    data_buffer1[i] = random[4 * i + 2] * box_width;
    data_buffer1[i + num_particles] = random[4 * i + 3] * box_height;
    data_buffer1[i + 2 * num_particles] = 1;

    data_buffer2[i + 2 * num_particles] = 1;    

/*    if (i == 0) {
      data_buffer1[i].z = 1000;
      data_buffer2[i].z = 1000;
      data_buffer1[i].x = box_width / 2;
      data_buffer1[i].y = box_height / 2;
      vels_buffer[i].x = 0;
      vels_buffer[i].y = 0;
    }
*/

    i += blockDim.x * gridDim.x;
  }
}

void alloc_data() {
  // instantiate particle_vels, particle_data on GPU
  gpuErrChk(hipMalloc((void **) &particle_vels[0], sizeof(float) * 2 * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_vels[1], sizeof(float) * 2 * num_particles));
  
  gpuErrChk(hipMalloc((void **) &particle_data[0], sizeof(float) * 3 * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_data[1], sizeof(float) * 3 * num_particles));

  gpuErrChk(hipMalloc((void **) &forces, 2 * sizeof(float) * num_particles));
}

void init_data(int h_num_particles, float box_width, float box_height, float min_vel, 
               float max_vel, int h_num_blocks, int h_num_threads_per_block) 
{
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  // instantiate particle_vels, particle_data on GPU
  alloc_data();
   
  // set initial values for particle_vels, particle_data on GPU
  float * random;
  gpuErrChk(hipMalloc((void **) &random, sizeof(float) * num_particles * 4));   
  
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniform(gen, random, num_particles * 4);

  cudaInitKernel<<<num_blocks, num_threads_per_block>>>(particle_vels[0], particle_data[0], particle_data[1], 
                                                        random, box_width, box_height, min_vel, max_vel, num_particles);

  hiprandDestroyGenerator(gen);
  gpuErrChk(hipFree(random));
}
void init_data(int h_num_particles, float *h_particle_data, float *h_particle_vels, int h_num_blocks, int h_num_threads_per_block) {
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  alloc_data();

  // TODO: Change this for memory coalescing
  gpuErrChk(hipMemcpy(particle_data[0], h_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_data[1], h_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_vels[0], h_particle_vels, 2 * num_particles * sizeof(float), hipMemcpyHostToDevice));
}

void delete_data() {
  // free all memory on GPU
  for (int i = 0; i < 2; i++)
  {
    gpuErrChk(hipFree(particle_vels[i]));
    gpuErrChk(hipFree(particle_data[i]));
  }

  gpuErrChk(hipFree(forces));
}

__device__
float2 get_force(float3 pos_data, float * data_old, int num_particles) {
  // sum force from every other particle based on mass, position of both particles
  float2 force = {0, 0};

  float3 other_data1; // saves about 3s @ 128 threads/block and 1024 particles to store data_old[i], x_dist, and y_dist locally
  float x_dist1, y_dist1;

  float force_magnitude1;
  for (int i = 0; i < num_particles; i++)
  {
    other_data1 = data_old[i];
    x_dist1 = pos_data.x - other_data1.x;
    y_dist1 = pos_data.y - other_data1.y;

    force_magnitude1 = pos_data.z * other_data1.z * pow(x_dist1 * x_dist1 + y_dist1 * y_dist1 + SOFT_FACTOR, -1.5f);

    force.x -= x_dist1 * force_magnitude1;
    force.y -= y_dist1 * force_magnitude1;
  }
  return force;
}

__global__
void calc_forces_kernel(float * forces, float * vels_old, float * vels_new, float * data_old, float * data_new, float dt, int num_particles) {
  extern __shared__ float sdata[];
  
  int tile_id = blockIdx.x;
  int tid = threadIdx.x;
  
  int num_tiles_per_col = num_particles / blockDim.x;
  int num_tiles = num_particles * num_particles / (blockDim.x * blockDim.x);

  float3 pos_data;
  
  pos_data.x = data_old[rid];
  pos_data.y = data_old[rid + num_particles];
  pos_data.z = data_old[rid + 2 * num_particles];

  while (tile_id < num_tiles)
  {
    int rid = (tile_id % num_tiles_per_col) * blockDim.x + tid;
    int cid = (tile_id/num_tiles_per_col) * blockDim.x + tid;

    sdata[tid] = data_old[cid];
    sdata[tid + blockDim.x] = data_old[cid + num_particles];
    sdata[tid + 2 * blockDim.x] = data_old[cid + 2 * num_particles];
 
    __syncthreads();

    float2 block_force = get_force(pos_data, sdata, blockDim.x);
    atomicAdd(forces + rid, block_force.x);
    atomicAdd(forces + rid + num_particles, block_force.y);
    
    __syncthreads();

    tile_id += gridDim.x;

    // Check if need to reload pos_data from global memory
    if (gridDim.x % num_tiles_per_col == 0) {
      pos_data.x = data_old[rid];
      pos_data.y = data_old[rid + num_particles];
      pos_data.z = data_old[rid + 2 * num_particles];
    }
  }
}

__global__
void apply_forces_kernel(float * forces, float * vels_old, float * vels_new, float * data_old, 
                         float * data_new, float dt, int num_particles)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  while (i < num_particles)
  {
    float2 force;
    force.x = forces[i];
    force.y = forces[i + num_particles];

    vels_new[i] = vels_old[i] + force.x * dt / data_old[i + 2 * num_particles]; // TODO: replace data_old[i] with pos_data
    vels_new[i + num_particles] = vels_old[i + num_particles] + force.y * dt / data_old[i + 2 * num_particles];
    
    data_new[i] = data_old[i] + vels_new[i] * dt; 
    data_new[i + num_particles] = data_old[i + num_particles] + vels_new[i + num_particles] * dt;

    i += blockDim.x * gridDim.x;
  }
}

void simulate_time_step(float dt) {
  // call kernel

  gpuErrChk(hipMemset(forces, 0, num_particles * 2 * sizeof(float)));

  calc_forces_kernel<<<num_blocks, num_threads_per_block, num_threads_per_block * sizeof(float) * 3>>>
                                                       (forces, particle_vels[pingpong], particle_vels[1 - pingpong], 
                                                         particle_data[pingpong], particle_data[1 - pingpong], 
                                                         dt, num_particles);
  
  apply_forces_kernel<<<num_blocks, num_threads_per_block>>>(forces, particle_vels[pingpong], particle_vels[1 - pingpong], 
                                                         particle_data[pingpong], particle_data[1 - pingpong], 
                                                         dt, num_particles);

  // update pingpong
  pingpong = 1 - pingpong;
}


void get_particle_data(float * h_particle_data, float * h_particle_vels) {
  // copy GPU data into particle_data, particle_vels array
  gpuErrChk(hipMemcpy(h_particle_data, particle_data[1 - pingpong], sizeof(float) * 3 * num_particles, hipMemcpyDeviceToHost));
  gpuErrChk(hipMemcpy(h_particle_vels, particle_vels[1 - pingpong], sizeof(float) * 2 * num_particles, hipMemcpyDeviceToHost));
}


std::string get_algorithm() {
  return std::string("PxP_OPT");
}