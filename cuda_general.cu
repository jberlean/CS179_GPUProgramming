#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <cstdio>
#include <iostream>

#include <hip/hip_runtime.h>

#include "n_body_sim_cuda.cuh"
#include "cuda_general.cuh"

// GENERAL UTILITY FUNCTIONS USED BY MULTIPLE ALGORITHMS

void alloc_particle_info(float3 **particle_data, float2 **particle_vels) {
  // instantiate particle_vels, particle_data on GPU
  gpuErrChk(hipMalloc((void **) &particle_vels[0], sizeof(float2) * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_vels[1], sizeof(float2) * num_particles));
  
  gpuErrChk(hipMalloc((void **) &particle_data[0], sizeof(float3) * num_particles));
  gpuErrChk(hipMalloc((void **) &particle_data[1], sizeof(float3) * num_particles));
}

__global__
void cudaInitKernel_uncoalesced(float2 * vels_buffer, float3 * data_buffer1, float3 * data_buffer2, float * random, float box_width, 
                    float box_height, float min_vel, float max_vel, int num_particles)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  while (i < num_particles)
  {
    vels_buffer[i].x = min_vel + random[4 * i] * (max_vel - min_vel);
    vels_buffer[i].y = min_vel + random[4 * i + 1] * (max_vel - min_vel);
    data_buffer1[i].x = random[4 * i + 2] * box_width;
    data_buffer1[i].y = random[4 * i + 3] * box_height;
    data_buffer1[i].z = 1;

    data_buffer2[i].z = 1;    

/*    if (i == 0) {
      data_buffer1[i].z = 1000;
      data_buffer2[i].z = 1000;
      data_buffer1[i].x = box_width / 2;
      data_buffer1[i].y = box_height / 2;
      vels_buffer[i].x = 0;
      vels_buffer[i].y = 0;
    }
*/

    i += blockDim.x * gridDim.x;
  }
}


void init_data_uncoalesced(int h_num_particles, float box_width, float box_height, float min_vel, 
               float max_vel, int h_num_blocks, int h_num_threads_per_block, float3 **particle_data, float2 **particle_vels) 
{
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  // instantiate particle_vels, particle_data on GPU
  alloc_particle_info();
   
  // set initial values for particle_vels, particle_data on GPU
  float * random;
  gpuErrChk(hipMalloc((void **) &random, sizeof(float) * num_particles * 4));   
  
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandGenerateUniform(gen, random, num_particles * 4);

  cudaInitKernel_uncoalesced<<<num_blocks, num_threads_per_block>>>(particle_vels[0], particle_data[0], particle_data[1], 
                                                        random, box_width, box_height, min_vel, max_vel, num_particles);

  hiprandDestroyGenerator(gen);
  gpuErrChk(hipFree(random));
}
void init_data_uncoalesced(int h_num_particles, int h_num_blocks, int h_num_threads_per_block,
    float *h_particle_data, float *h_particle_vels, float3 **particle_data, float2 **particle_vels) {
  num_particles = h_num_particles;
  num_blocks = h_num_blocks;
  num_threads_per_block = h_num_threads_per_block;

  alloc_particle_info();

  gpuErrChk(hipMemcpy(particle_data[0], h_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_data[1], h_particle_data, 3 * num_particles * sizeof(float), hipMemcpyHostToDevice));
  gpuErrChk(hipMemcpy(particle_vels[0], h_particle_vels, 2 * num_particles * sizeof(float), hipMemcpyHostToDevice));
}

void delete_data_uncoalesced(float **particle_data, float **particle_vels) {
  // free all memory on GPU
  for (int i = 0; i < 2; i++)
  {
    gpuErrChk(hipFree(particle_vels[i]));
    gpuErrChk(hipFree(particle_data[i]));
  }
}

void get_particle_data_uncoalesced(float * h_particle_data, float * h_particle_vels, float3 **particle_data, float2 **particle_vels) {
  // copy GPU data into particle_data, particle_vels array
  gpuErrChk(hipMemcpy(h_particle_data, particle_data[1 - pingpong], sizeof(float) * 3 * num_particles, hipMemcpyDeviceToHost));
  gpuErrChk(hipMemcpy(h_particle_vels, particle_vels[1 - pingpong], sizeof(float) * 2 * num_particles, hipMemcpyDeviceToHost));
}
